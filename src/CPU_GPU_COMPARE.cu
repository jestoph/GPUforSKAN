#include "hip/hip_runtime.h"
/*



*/


// Standard Libraries
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<sys/time.h> /*gettimeofday*/

// Includes the cuda runtime routines
#include<hip/hip_runtime.h>


//TODO: These should happily be .h files, but this fails for some reason....

// My preprocessor functions
#include "CPU_GPU.h"

// SKAN_kernel 
#include "SKAN_KERNEL.h"
#include "multiNeuron.h"

// Some Helper device Functions
#include "deviceFunctions.h"

// CPU SKAN
#include "SKAN_CPU.h"

// CSV read/write operations
#include "CSV_IO.h"

// Function to create a random input
#include "createInput.h"




#define TEST 1
#define VIEWRESULT 1

/*
*
* Function Prototypes
*
*/

/*GPU SKAN Functions*/
//__global__ void GPU_SKAN_KERNEL(float * input,  float  * dendriteValue, int timeLength);
float * GPU_SKAN(int channels, int timeLength, float * h_input, int reps);

/*CPU Skan Functions*/
float * CPU_SKAN(int channels, int timeLength, float * input, int reps);

/*CSV I/O Functions*/
//void csvWrite(char * name,  int height, int length, float * vector);
//float * csvRead(char * name);

/* Function to make a random input */
float * createInput(int channels,int vecLength);

/**************************************************************************************************************
		MAIN
	
	Handle input, send out to the cpu and gpu functions 
	So it can easily be viewed by profiler
**************************************************************************************************************/
int main(int argc, char * argv[]){

	DEBUG_MSG;

	// Create Context so that this doesn't get measured
	// Otherwise this will add around 30 - 50 mS of execution time to the 
	// Initial API call
	hipFree(0);
	srand(time(NULL) ); // Seed the RNG for the input generation
	/*LOAD UP ARRAY LENGTH*/

	int reps = 1;
	int vecLength = 1000000;//VECLENGTH;
	int channels = 32;//CHANNELS;
	int numberofWarps = 32;

	/*ALL GOOD, LETS GET ON WITH IT*/ 
	printf("\tRunning %d repetitions with %d channels and %d timesteps\n",reps, channels,vecLength);

	// Create an input - NEED TO FREE
	//printf("\tCreate input\n");
	float * input = createInput(channels,vecLength);
	float multiNeuron_cpu_time;
	float * output;
	CPU_TIMER(output = multiNeuron_hostFn(vecLength,input,reps, numberofWarps),multiNeuron_cpu_time);
	printf("\tMultineuron took %f\n",(double)multiNeuron_cpu_time/(double)CLOCKS_PER_SEC);

	char writeOut[] = "data/OUTPUT.csv";
	//measure the time taken to write out the output
	float writeouttime;
	CPU_TIMER(csvWrite(writeOut,channels + numberofWarps,vecLength,output),writeouttime);
	printf("output took %f seconds to write out and is %d bytes not including commas and newlines\n",(double)writeouttime/(double)CLOCKS_PER_SEC, (channels + numberofWarps) * vecLength *sizeof(float));

	char csv_input[] = "data/INPUT.csv";
	CPU_TIMER(csvWrite(csv_input,channels, vecLength,input),writeouttime);
	printf("input took %f seconds to write out and is %d bytes not including commas and newlines\n",(double)writeouttime/(double)CLOCKS_PER_SEC, (channels) * vecLength *sizeof(float));

	free(input);
	free(output);



	return 0;

	/****************************************
	* WE NOW HAVE VALID INPUT, APPLY KERNNELS
	*****************************************/

	//for(int reps = 1 ; reps < 1000 ; reps ++){
	// Is a float, returns the whole array so need to be sure to free the memory	
	//printf("\tApplying GPU Kernel to input\n");	

	/**************************************
	* DO PROCESSING, TIME THE PROCESS
	***************************************/	
	//float CPU_clock_time,GPU_clock_time,
	float CPU_cpu_time, GPU_cpu_time;
	float * result_GPU, * result_CPU;


	printf("\tCPU...");
	CPU_TIMER(result_CPU = CPU_SKAN(channels, vecLength, input, reps), CPU_cpu_time);
	printf("Done.\n\tGPU...");
	CPU_TIMER(result_GPU = GPU_SKAN(channels, vecLength, input, reps), GPU_cpu_time);
	printf("Done.\n");

	/*Run on GPU, time with CPU timer*/
	//CLOCK_TIMER_uSEC(result_GPU = GPU_SKAN(channels, vecLength, input, reps), GPU_clock_time);

	/*Run on CPU, time wirh CPU timer*/
	//CLOCK_TIMER_uSEC(result_CPU = CPU_SKAN(channels, vecLength, input, reps), CPU_clock_time);

	printf("\tCPU_cpu %lf, GPU_cpu %lf\n",(double)CPU_cpu_time/(double)CLOCKS_PER_SEC,(double)GPU_cpu_time/(double)CLOCKS_PER_SEC);



	//printf("\tCPU_clock %lf GPU_clock %lf\n",CPU_clock_time,GPU_clock_time),;

	//printf("\tDifference_clock %lf Difference_cputime %lf\n",CPU_clock_time - GPU_clock_time,(double)(CPU_cpu_time-GPU_cpu_time)/(double)CLOCKS_PER_SEC);

	/*
	*
	*
	*
	*
	*
	*
	*
	*
	*/

	/****************************************
	* PROCESSING DONE, CHECK RESULT
	****************************************/

	float FLOATERR = 0.000001;
	float maxError = FLOATERR, temp;
	int maxErrorTimestep = 0, maxErrorHeight =0;

	for(int i = 0 ; i < (channels + reps)*vecLength; i++){ 

		temp = abs(result_CPU[i] - result_GPU[i]);
		if (temp > maxError){
			//printf("\t!error: result_CPU[%d] = %f, result_GPU[%d] = %f\n",i,result_CPU[i],i,result_GPU[i]);
			maxErrorTimestep = i%vecLength; //mod returns the remainder
			maxErrorHeight = i/vecLength; // integer arithmetic returns discards remainder
			maxError = temp;
		}
	}

	if(maxError>FLOATERR)printf("\tMaxerror = %f at height %d and timestep %d\n",maxError,maxErrorHeight,maxErrorTimestep);
	else printf("\tThe two outputs are identical\n");

#if 1 
	if(VIEWRESULT){	
		printf("\tWriting the Input, and the CPU and GPU outputs to .csv files\n");
		char csv_input[] = "data/INPUT.csv";
		char csv_CPU[] = "data/CPU.csv";
		char csv_GPU[] = "data/GPU.csv";

		csvWrite(csv_input,channels, vecLength,input);
		csvWrite(csv_CPU,channels + reps, vecLength,result_CPU);
		csvWrite(csv_GPU,channels + reps,vecLength,result_GPU);
	}
	
	free(result_CPU);
	free(result_GPU);
	free(input);
#endif

	
	return 0;
	
}
