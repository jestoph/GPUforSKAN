#include "hip/hip_runtime.h"


#include "includes.h"

#include "deviceFunctions.h"

/**********************************************************************************************************************************
*  GPU KERNEL
*
*  Mostly taken from example code
*
*`	TODO: do something about the cruft at the top of the function.
*
*
*
*
************************************************************************************************************************************/


// what does const mean in this context? is it putting into global memory?
__global__ void GPU_SKAN_KERNEL(float * input,  float  * dendriteValue, int timeLength, int nextPowerof2, int numberOfBlocks){
/*****REMOVE ALL THIS SHIT*****/
int chs = blockDim.x;
int T=400; // T = number of timesteps between presentations?
int C=4; // ratio between dr_max/dr_min
//    %%% Synapse-Dendrite Paramters

int rampMax = 10000;  //% maximum value r can have aka synaptic weight (w) in paper
int dRampSlope = 1; //%(ddr) in paper
float rampSlope_min=2*(2*C)/(float)(2*C-1)*rampMax/(float)T;
//printf("rampslopemin = %f\n",rampSlope_min);
float rampSlope_max=C*rampSlope_min; //% del_r_max in paper
   
//    %%% Soma Parameters
float threshIncrease =.004*rampMax*chs; //% INCREASE in thresh during firing should be about half threshDecay
float postFiringThreshDecay = .01*rampMax*chs;//% threshIncrease*2; % 'forgetting' parameter
float noFiringThreshDecay =.01*rampMax*chs;
float threshold=.5*rampMax*chs;

//    %%% Layer parameters
float inhibitionMax=rampMax;
float inhibitionDecay=2*inhibitionMax/T;






/******************************/
	int blockSize = blockDim.x;
	int stride = blockDim.x + numberOfBlocks;

	// allocate shared memory
	extern __shared__ float sReduce[];
	int offset  = threadIdx.x;

	/* Dendritic parameters and logic structures */
	int edgecase = 0; // this is to handle if the value goes below zero
	int somaOutput = 0;

	int logicOffset = 1; // in order to map -1 to 0
	int CHANGESTATE[3];

	float rampSlope = 0;// doesn't matter if 0 will be clamped anyway.
	//float rampmax = 10000;

	// Set up the state machine - states defined in preprocesser	
	int CURRENTSTATE = NOTHING;

	/* Global Parameters*/
	/*TODO: 	think about using preprocessor to get rid of some of these values? 
			How many variables can the kernel hold without having to store them in cache? - ans: about 32*/
	float somaSum = 0;
	float somaSumPrev = 0; // <- AAARGH IT WAS THIS BASTARD!

	/* Global Flags - Dont need some of these because of the K=0 thing*/
	//int JUSTSTOPPED = 0;
	int SPIKING = 0;
	//int NOFIREEVENT = 0;	

	int inhibition = 0;//INHIBITION_INITIAL;
	int inhibitionPrev=0;



	/* Run the Simulation*/

	dendriteValue[threadIdx.x] = 0;// 0;//threadIdx.x;
	dendriteValue[blockSize + blockIdx.x] = 0;//threshold;//0;
	float temp = 0; /*temp to avoid global memory access*/	

	for(int time = 1 ; time < timeLength ; time++){

		/*
		* GLOBAL DATA
		* Update Rampmax, Rampslope and wInputFlag depending on the global flags
		* Using boolean flags for arithmetic to avoid branching.
		*/

		inhibition = fmaxf(inhibition - inhibitionDecay,0);

		// All this shit here can get fucked.
		//if(JUSTSTOPPED){
		//	rampMax += wOutputAfterInputRise * wInputFlag - wOutputWithoutInputFall * (~wInputFlag);
		//	wInputFlag = 0;
		//}
		//else
		if(SPIKING){
			rampSlope += CURRENTSTATE*dRampSlope;
		}
		//else if(NOFIREEVENT){
		//	rampMax -= winputwithoutoutputfall * wInputFlag;
		//	wInputFlag = 0;
		//}


		/* Clamp values to ensure stability*/
		/* TODO: these can be put in the appropriate 'if' statement to avoid evaluating them when nothing's happening*/
		//rampMax = clamp_branch(rampMax, wUpperBound,wLowerBound);
		rampSlope = clamp(rampSlope,rampSlope_max,rampSlope_min);


		/*
		* LOCAL, DENDRITIC DATA
		* Change the dendritic state
		* Take the sum and store back to the array
		*/


		// Move from nothing if there is an input - no need to check state, that is done later
		CHANGESTATE[NOTHING + logicOffset] = NEXTSTATE(NOTHING,GOINGUP, (input[time * blockSize + offset]));
		// Move from ramping up to ramping down if you have exceeded rampMax
		CHANGESTATE[GOINGUP + logicOffset] = NEXTSTATE(GOINGUP,GOINGDOWN, (temp > rampMax));
		// Move from ramping down to nothing if you go below zero or are zero
		CHANGESTATE[GOINGDOWN + logicOffset] = NEXTSTATE(GOINGDOWN,NOTHING, (temp<=0 ));

		// Handles if the output goes below zero, in order for it to come back up
		edgecase = temp < 0;

		// Update the state machine - this is where all the madness starts to make sense
		CURRENTSTATE = CHANGESTATE[CURRENTSTATE + logicOffset];


		// Now can use the state (which is actually 1 or 0 or -1) to multiply the dendrite
		temp = (1 - edgecase) * temp + CURRENTSTATE * rampSlope;


		sReduce[ threadIdx.x] =	temp;
		
		// Use this placeholder variable so we don't need lots of memory access
		somaSum = reduce( sReduce,  blockSize,  nextPowerof2);
		somaSum = sReduce[0]; // <- WITHOUT THIS LINE EVERYTHING BREAKS


		/*UPDATE THE GLOBAL STATE*/

		// Set to zero so can use if--elseif--elseif to avoid extra evaluation
		SPIKING = 0;
		//JUSTSTOPPED = 0;
		//NOFIREEVENT = 0;

		/* FIRING */
            	if ((inhibitionPrev == 0 ||  somaOutput == 1) && somaSum>=threshold){
			SPIKING = 1;
			somaOutput = 1;
			threshold += threshIncrease;// THRESHINCREASE; 
			inhibition = inhibitionMax;// INHIBITION_MAX;
		}
		/* JUST STOPPED */
		else if(somaOutput == 1 && somaSum < threshold){
			//JUSTSTOPPED = 1;
			somaOutput = 0;
			threshold -= postFiringThreshDecay;//  THRESHDECAY_POSTFIRE;
		}
		/* THERE WAS A NO FIRING EVENT */
            	else if( somaSumPrev > 0 && somaSum <= 0 && inhibitionPrev == 0){ // if  rsum just hit zero.(\_) this spike series event is over
			//NOFIREEVENT = 1;
			somaSum = 0; // <- incase it went below zero
			somaOutput = 0;
			threshold -=  noFiringThreshDecay;// THRESHDECAY_NOFIRE;
		}
		// This is incase there was a no-firing event
		somaSumPrev = somaSum;
		inhibitionPrev = inhibition;

		/*
		* Store the Results
		* Get thread 0 to do this so there is no concurrency problem
		* This WILL diverge the warp, but it's necessary.
		*/
		dendriteValue[time*stride + threadIdx.x] = temp;// sReduce has already been changed
		if(threadIdx.x == 0 ){
			// Store the Soma Sum
			dendriteValue[time*stride + blockSize + blockIdx.x] = SPIKING;//inhibition;//threshold;//somaSum;//SPIKING ;//somaSum;
			// Store the Soma Output
			
		}
	}


}

/************************************************************************************************************************
		GPU SKAN

	Host-side code to manage memory allocation and transfer and call the kernel

CURRENTLY ONLY ABLE TO BE SINGLE NEURON, MAINLY JUST WANT TO CHECK FUNCTIONALITY

************************************************************************************************************************/

float * GPU_SKAN(int channels, int timeLength, float * h_input, int reps){

	// Allocate space for input and output on the device, copy data over
	float * d_input, * d_dendriteValues;
	CUDAMALLOC(d_input, channels * timeLength);
	CUDAMALLOC(d_dendriteValues, (channels + reps) * timeLength);
	CUDAMEMCPY_H2D(h_input,d_input, channels * timeLength);

	// find the next lowest power of two
	int nextPowerof2 = 1;
	int temp = channels;
	while((temp>>=1)){
		nextPowerof2<<=1;
	}


	//GPU_SKAN_KERNEL<<<numberOfBlocks, threadsPerBlock, sharedmemory>>>(params);
	//GPU_SKAN_KERNEL_old<<<reps, channels, channels*sizeof(float)>>>(d_input,d_dendriteValues,timeLength,nextPowerof2);	

	GPU_SKAN_KERNEL<<<reps, channels, channels*sizeof(float)>>>(d_input,d_dendriteValues,timeLength,nextPowerof2,reps);	
	
	//hipDeviceSynchronize(); // block until the device finishes previous call

	// Allocate space for the output on the host
	float * h_dendriteValues;
	HOSTMALLOC(h_dendriteValues, (channels + reps) * timeLength);

	// Copy result back to host
	CUDAMEMCPY_D2H(d_dendriteValues, h_dendriteValues, (channels + reps) * timeLength);

/*	
	// Free device memory
	CUDAFREE(d_input);
	CUDAFREE(d_dendriteValues);

	// Now it is polite to reset the device
	CUDACLEANUP();
*/
	return h_dendriteValues;
}



