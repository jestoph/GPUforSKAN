#include "hip/hip_runtime.h"
#include "deviceFunctions.h"
#include <stdio.h>

#ifndef DEVICEFUNCTIONS_CU
#define DEVICEFUNCTIONS_CU

/****************************************************************************
*	My clamp function
*	USAGE clamp(value, max,min)
*
*	Avoids branching by using intrinsic max/min functions
******************************************************************************/
__device__ float clamp(float value,float max, float min){
	// is it intrinsic? http://llvm.org/docs/LangRef.html#fcmp-instruction and http://llvm.org/docs/LangRef.html#select-instruction
	// select is like cond?a:b 'without IR-level branching.' meaning intermediate representation
	// could do: float maxRes[2] = {a,b}
	//		result = maxRes[a>b]; select <cond> <trueval> <falseval> nobranch!
	/*
	*	look for
	*
	*/
	// fcmp
	return(fminf(fmaxf(value,min),max));
}

/***************************************************************************
*
*	A Warp-divergent Clamp function
*	Just to demonstrate the inefficiency
*
***************************************************************************/
__device__ float clamp_branch(float value,float max,float min){

	if(value>max) value = max;
	else if(value<min) value = min;

	return value;
}

/******************************************************************************
*	My Reduction Function
*	Works for any number of inputs
*
*	This function will actually be inlined after compilation
********************************************************************************/
__device__ float reduce( float * sReduce, int blockSize, int nextPowerof2){

	/************************************************************************
	 *	Take the sum of the dendrites, store in the channels + 1 column
	 *	DO NOT MESS WITH THIS LOGIC - IT WORKS!!!!
	 ************************************************************************/

	int difference = blockSize - nextPowerof2;
	
	//Ensure previous load is completed
	__syncthreads();

	// handle the non-power of two bits
	if(difference && threadIdx.x < difference){
		sReduce[threadIdx.x] += sReduce[threadIdx.x + nextPowerof2];
	}

	// Make sure load is completed
	__syncthreads();

	// This is the normal, naive reduction for 2^N threads
	for(int n = nextPowerof2>>1; n>=1; n>>=1){
	
		if(threadIdx.x < n)
			sReduce[threadIdx.x] += sReduce[threadIdx.x + n];
	
		__syncthreads();
	}
	// result will be stored in dendriteValue[time*stride + blockSize]
		
	return( sReduce[threadIdx.x] );


}

__device__ float betterReduce( float * sReduce, int blockSize, int nextPowerof2){

	/************************************************************************
	 *	Take the sum of the dendrites, store in the channels + 1 column
	 *	DO NOT MESS WITH THIS LOGIC - IT WORKS!!!!
	 ************************************************************************/

	int difference = blockSize - nextPowerof2;
	int tid = threadIdx.x;
	//Ensure previous load is completed
	__syncthreads();

	// handle the non-power of two bits
	if(difference && threadIdx.x < difference){
		sReduce[threadIdx.x] += sReduce[threadIdx.x + nextPowerof2];
		//__syncthreads; // This is not necessary because there is no collision 
	}

	
	// Conditionally reduce for different blocksizes
	if (blockSize >= 512) {if (tid < 256) { sReduce[tid] += sReduce[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) {if (tid < 128) { sReduce[tid] += sReduce[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) {if (tid < 64) { sReduce[tid] += sReduce[tid + 64]; } __syncthreads(); }

	// We are at the final warp, no longer need to sync threads.
	// these 'if blocksize' should be evaluated at compile time, so shouldn't be slow.
	if (tid < 32){
		if (blockSize >= 64) sReduce[tid] += sReduce[tid + 32];
		if (blockSize >= 32) sReduce[tid] += sReduce[tid + 16];
		if (blockSize >= 16) sReduce[tid] += sReduce[tid + 8];
		if (blockSize >= 8) sReduce[tid] += sReduce[tid + 4];
		if (blockSize >= 4) sReduce[tid] += sReduce[tid + 2];
		if (blockSize >= 2) sReduce[tid] += sReduce[tid + 1];
	}		

	return( sReduce[0] );


}

/****************************************************************
*
*	Warp Reduce - works for 32 Warps of 32 Threads ONLY
*	__syncthreads not necessary as it calculates within warps which execute in parallel
*	INTERESTING RESULT: SYNCTHREADS() STILL NECESSARY EVEN WITHIN WARP! IS THIS BECAUSE OF SHARED MEMORY LATENCY?:w
*	http://forums.udacity.com/questions/100037602/why-__syncthreads-needed-even-within-the-warp
*
*	Fixed this by using a volatile float. Note this may break in future.
*	
****************************************************************/
__device__ float warpReduce(volatile float * sReduce, int thisWarp){


//	TODO: don't need the ifs here, can just toss away useless results in non-thisWarp threads


	if(threadIdx.x < blockDim.x - 16){
		sReduce[threadIdx.x] += sReduce[threadIdx.x + 16];
		//__syncthreads(); // not sure if this is necessary
		 sReduce[threadIdx.x] += sReduce[threadIdx.x + 8];
		//__syncthreads(); // not sure if this is necessary
		 sReduce[threadIdx.x] += sReduce[threadIdx.x + 4];
		//__syncthreads(); // not sure if this is necessary
		 sReduce[threadIdx.x] += sReduce[threadIdx.x + 2];
		//__syncthreads(); // not sure if this is necessary
		sReduce[threadIdx.x] += sReduce[threadIdx.x + 1];
		//__syncthreads(); // not sure if this is necessary
	}

	// Result is now in sReduce[thisWarp]

	return (sReduce[thisWarp]);
	



}

#endif
