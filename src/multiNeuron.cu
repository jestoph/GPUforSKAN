#include "hip/hip_runtime.h"

#include<hip/hip_runtime.h>
#include "CPU_GPU.h"
#include <stdio.h>
#include <stdlib.h>
#include "deviceFunctions.h"


#define GLOBALINHIBITION

/**********************************************************************************************************************************
*  GPU KERNEL
*
*  Mostly taken from example code
*
*`	TODO: do something about the cruft at the top of the function.
*
*
*
*
************************************************************************************************************************************/


// what does const mean in this context? is it putting into global memory?
__global__ void multiNeuron(float * input,  float  * dendriteValue, int timeLength, int numberofWarps){

	// Memory for reduction
	// This has been declared volatile for warp synchronisation
	// Will be deprecated for devices over compute 2.3
	volatile extern  __shared__ float sReduce[];

	// Some local variables for atomic operations
	__shared__ int globalInhibition;
	globalInhibition = 0;

	// A lock that determines if this thread is allowed to edit inhibition
	volatile __shared__ int inhibitionLock;
	inhibitionLock = -1; // set to an invalid threadId to start with 


	// find the thread and memory offset for global memory
	int warpMemoryOffset = threadIdx.x/warpSize;

	// Each warp is identified by its lowest thread ie threadIdx.x = 0,32,64,96,128...
	int thisWarp = warpSize * warpMemoryOffset; 

	int inputStride = warpSize;
	int outputStride = warpSize + numberofWarps;


/*****REMOVE ALL THIS SHIT*****/
int chs = warpSize;
int T=400; // T = number of timesteps between presentations?
int C=4; // ratio between dr_max/dr_min
//    %%% Synapse-Dendrite Paramters

int rampMax = 10000;  //% maximum value r can have aka synaptic weight (w) in paper
int dRampSlope = 1; //%(ddr) in paper
float rampSlope_min=2*(2*C)/(float)(2*C-1)*rampMax/(float)T;
//printf("rampslopemin = %f\n",rampSlope_min);
float rampSlope_max=C*rampSlope_min; //% del_r_max in paper
   
//    %%% Soma Parameters
float threshIncrease =.004*rampMax*chs; //% INCREASE in thresh during firing should be about half threshDecay
float postFiringThreshDecay = .01*rampMax*chs;//% threshIncrease*2; % 'forgetting' parameter
float noFiringThreshDecay =.01*rampMax*chs;
float threshold=.5*rampMax*chs;

//    %%% Layer parameters
float inhibitionMax=rampMax;
float inhibitionDecay=2*inhibitionMax/T;
/******************************/

	int edgecase = 0; // this is to handle if the value goes below zero
	int somaOutput = 0;

	int logicOffset = 1; // in order to map -1 to 0
	int CHANGESTATE[3];

	float rampSlope = 0;// doesn't matter if 0 will be clamped anyway.
	//float rampmax = 10000;

	// Set up the state machine - states defined in preprocesser	
	int CURRENTSTATE = NOTHING;

	/* Global Parameters*/
	/*TODO: 	think about using preprocessor to get rid of some of these values? 
			How many variables can the kernel hold without having to store them in cache? - ans: about 32*/
	float somaSum = 0;
	float somaSumPrev = 0; // <- AAARGH IT WAS THIS BASTARD!

	/* Global Flags - Dont need some of these because of the K=0 thing*/
	//int JUSTSTOPPED = 0;
	int SPIKING = 0;
	//int NOFIREEVENT = 0;	

	int localInhibition = 0;//INHIBITION_INITIAL;
	int inhibitionPrev=0;



	/* Run the Simulation*/

	// Initial Conditions
	if(threadIdx.x < warpSize){
		dendriteValue[threadIdx.x] = 0;// 0;//threadIdx.x;
		dendriteValue[warpSize + threadIdx.x] = 0;//threshold;//0;
	}
	float temp = 0; /*temp to avoid global memory access*/	

//	if(threadIdx.x == thisWarp) printf("I am %3d, I own warp %3d and my final result is %2.2f\n",threadIdx.x, warpMemoryOffset,sReduce[thisWarp]);
	for(int time = 1 ; time < timeLength; time ++){


/***********************************copied code*****************************************/

		/* Taken word for word from the SKAN_KERNEL.cu implementation*/
		// Does my warp own the inhibtion lock? If so, decay. If it hits zero, give up the lock
		if(inhibitionLock == threadIdx.x){
			globalInhibition = fmaxf(globalInhibition - inhibitionDecay,0);
			if(globalInhibition == 0) inhibitionLock = -1;
		}
		

		if(SPIKING){
			rampSlope += CURRENTSTATE*dRampSlope;
		}

		rampSlope = clamp(rampSlope,rampSlope_max,rampSlope_min);

		/* LOCAL, DENDRITIC DATA */

		// Move from nothing if there is an input - no need to check state, that is done later
		CHANGESTATE[NOTHING + logicOffset] = NEXTSTATE(NOTHING,GOINGUP, (input[time * inputStride + threadIdx.x - thisWarp]) );
		// Move from ramping up to ramping down if you have exceeded rampMax
		CHANGESTATE[GOINGUP + logicOffset] = NEXTSTATE(GOINGUP,GOINGDOWN, (temp > rampMax));
		// Move from ramping down to nothing if you go below zero or are zero
		CHANGESTATE[GOINGDOWN + logicOffset] = NEXTSTATE(GOINGDOWN,NOTHING, (temp<=0 ));

		// Handles if the output goes below zero, in order for it to come back up
		edgecase = temp < 0;

		// Update the state machine - this is where all the madness starts to make sense
		CURRENTSTATE = CHANGESTATE[CURRENTSTATE + logicOffset];


		// Now can use the state (which is actually 1 or 0 or -1) to multiply the dendrite
		temp = (1 - edgecase) * temp + CURRENTSTATE * rampSlope;

		sReduce[ threadIdx.x] =	temp;
		
		// Use this placeholder variable so we don't need lots of memory access
		somaSum = warpReduce( sReduce, thisWarp);  // DIFFERENT FROM KERNEL CODE
		somaSum = sReduce[thisWarp]; // DIFFERENT FROM KERNEL CODE


		/*UPDATE THE GLOBAL STATE*/

		// Set to zero so can use if--elseif--elseif to avoid extra evaluation
		SPIKING = 0;
		/* FIRING */
		// This has been moved around so that the atomicCAS isn't called unless it needs to be.
            	if ((inhibitionPrev == 0 || somaOutput == 1) && somaSum>=threshold){ // DIFFERENT FROM KERNEL CODE

			// First thread that gets here gets the lock!
			// Lazy evaluation means it will only do atomic swap once per firing.
			if(somaOutput == 1 || atomicCAS((int *)&globalInhibition,0,1)==0) {inhibitionLock = thisWarp;}
			if(inhibitionLock == thisWarp){
				SPIKING = 1;
				somaOutput = 1;
				threshold += threshIncrease;// THRESHINCREASE; 
				globalInhibition = inhibitionMax;// INHIBITION_MAX;
			}
		}
		/* JUST STOPPED */
		else if(somaOutput == 1 && somaSum < threshold){
			// Set the global inhibition signal back to zero
			somaOutput = 0;
			threshold -= postFiringThreshDecay;//  THRESHDECAY_POSTFIRE;
		}
		/* THERE WAS A NO FIRING EVENT */
            	else if( somaSumPrev > 0 && somaSum <= 0 && inhibitionPrev == 0){ // if  rsum just hit zero.(\_) this spike series event is over
			somaSum = 0; // <- incase it went below zero
			somaOutput = 0;
			threshold -=  noFiringThreshDecay;// THRESHDECAY_NOFIRE;
		}
		// This is incase there was a no-firing event
		somaSumPrev = somaSum;

		__syncthreads(); // <-- THIS IS CRUCIAL
		inhibitionPrev = globalInhibition;//localInhibition;



/***************************end copied code******************************************/

		// Just store the first warp's results
		if(threadIdx.x < warpSize) dendriteValue[ time * outputStride + threadIdx.x] = temp;// sReduce has already been changed
		if(threadIdx.x == thisWarp ){
			// Store the Soma Sum
			dendriteValue[time*outputStride + warpSize + warpMemoryOffset] = somaOutput;//lobalInhibition;//somaOutput;//SPIKING;//threshold;//somaSum;//SPIKING ;//somaSum;
			//dendriteValue[time*outputStride + 31] = threshold;
			//dendriteValue[time*outputStride + 30] = somaSum;
			// Store the Soma Output
			
		}

	}

}

/************************************************************************************************************************
		GPU SKAN

	Host-side code to manage memory allocation and transfer and call the kernel

CURRENTLY ONLY ABLE TO BE SINGLE NEURON, MAINLY JUST WANT TO CHECK FUNCTIONALITY

************************************************************************************************************************/

float * multiNeuron_hostFn(int timeLength, float * h_input, int reps, int numberofWarps){

	// Allocate space for input and output on the device, copy data over
	float * d_input, * d_dendriteValues; 
	int channels = 32;

	CUDAMALLOC(d_input, channels * timeLength);
	CUDAMALLOC(d_dendriteValues, (numberofWarps + channels)* timeLength);
	CUDAMEMCPY_H2D(h_input,d_input, channels * timeLength);


	multiNeuron<<<reps, 32*numberofWarps, 32*numberofWarps*sizeof(float)>>>(d_input, d_dendriteValues, timeLength,numberofWarps);	

	hipDeviceSynchronize(); // block until the device finishes previous call

	// Allocate space for the output on the host
	float * h_dendriteValues;
	HOSTMALLOC(h_dendriteValues, 2*channels* timeLength);

	// Copy result back to host
	CUDAMEMCPY_D2H(d_dendriteValues, h_dendriteValues, (numberofWarps + channels) * timeLength);

	// Free device memory
	//CUDAFREE(d_input);
	//CUDAFREE(d_dendriteValues);

	// Now it is polite to reset the device
	//CUDACLEANUP();

	return h_dendriteValues;

}


